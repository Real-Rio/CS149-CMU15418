#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256

// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n)
{
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result
__global__ void upsweep(int two_dplus1, int two_d, int N, int *output)
{
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int i = thread_id * two_dplus1;
    if (i < N)
    {
        output[i + two_dplus1 - 1] += output[i + two_d - 1];
    }
}

__global__ void downsweep(int two_dplus1, int two_d, int N, int *output)
{
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int i = thread_id * two_dplus1;
    if (i < N)
    {
        int t = output[i + two_d - 1];
        output[i + two_d - 1] = output[i + two_dplus1 - 1];
        output[i + two_dplus1 - 1] += t;
    }
}

__global__ void setElementToZero(int *output, int N, int idx)
{
    output[idx] = 0;
}

void exclusive_scan(int *input, int N, int *result)
{
    int two_dplus1;
    int num;
    int blocks;

    // N = nextPow2(N);
    // CS149 TODO:
    //
    // Implement your exclusive scan implementation here.  Keep in
    // mind that although the arguments to this function are device
    // allocated arrays, this is a function that is running in a thread
    // on the CPU.  Your implementation will need to make multiple calls
    // to CUDA kernel functions (that you must write) to implement the
    // scan.
    for (int two_d = 1; two_d <= N / 2; two_d *= 2)
    {
        two_dplus1 = two_d * 2;
        num = (N + two_dplus1 - 1) / two_dplus1;
        blocks = (num + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

        if (num < THREADS_PER_BLOCK)
        {
            upsweep<<<1, num>>>(two_dplus1, two_d, N, result);
        }
        else
        {
            upsweep<<<blocks, THREADS_PER_BLOCK>>>(two_dplus1, two_d, N, result);
        }
        hipDeviceSynchronize();
    }

    setElementToZero<<<1, 1>>>(result, N, N - 1);
    hipDeviceSynchronize();

    for (int two_d = N / 2; two_d >= 1; two_d /= 2)
    {
        two_dplus1 = two_d * 2;
        num = (N + two_dplus1 - 1) / two_dplus1;
        blocks = (num + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

        if (num < THREADS_PER_BLOCK)
        {
            downsweep<<<1, num>>>(two_dplus1, two_d, N, result);
        }
        else
        {
            downsweep<<<blocks, THREADS_PER_BLOCK>>>(two_dplus1, two_d, N, result);
        }
        hipDeviceSynchronize();
    }

    setElementToZero<<<1, 1>>>(result, N, 0);
}

//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int *inarray, int *end, int *resultarray)
{
    int *device_result;
    int *device_input;
    int N = end - inarray;

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);

    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, N * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, rounded_length, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, device_result, N * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;
    return overallDuration;
}

// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int *inarray, int *end, int *resultarray)
{

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);

    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration;
}

// Computes bits indicating repeats and their indices.
//
// Note that the out_idxs array will simply have the same set of values as the
// flags array at this point, but we will apply exclusive_scan afterwards.
__global__ void find_repeat_flags(int N, int *device_input, int *flags,
                                  int *out_idxs)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N - 1)
    { // Skip the last element.
        int flag_i = 0;
        if (device_input[index] == device_input[index + 1])
        {
            flag_i = 1;
        }
        flags[index] = flag_i;
        out_idxs[index] = flag_i;
    }
}

// Finds repeated elements and adds them to the output array.
__global__ void find_repeat_elements(int N,
                                     int *flags,
                                     int *out_idxs,
                                     int *device_output)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N - 1 && flags[index] == 1)
    { // Skip the last element.
        const int out_idx = out_idxs[index];
        device_output[out_idx] = index;
    }
}

// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
int find_repeats(int *device_input, int length, int *device_output)
{
    // CS149 TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.
    const int blocks = (length + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    const int N = nextPow2(length);

    int *flags = nullptr;
    int *out_idxs = nullptr;
    hipMalloc((void **)&flags, N * sizeof(int));
    hipMalloc((void **)&out_idxs, N * sizeof(int));
    find_repeat_flags<<<blocks, THREADS_PER_BLOCK>>>(length, device_input, flags,
                                                     out_idxs);
    hipDeviceSynchronize();

    exclusive_scan(flags, N, out_idxs);
    hipDeviceSynchronize();

    find_repeat_elements<<<blocks, THREADS_PER_BLOCK>>>(length,
                                                        flags, out_idxs,
                                                        device_output);
    int *num_ret = new int[1];
    hipMemcpy(num_ret, out_idxs + length - 1, 1 * sizeof(int), hipMemcpyDeviceToHost);
    const int ret = num_ret[0];
    delete[] num_ret;

    hipDeviceSynchronize();

    hipFree(flags);
    hipFree(out_idxs);

    return ret;
}

//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length)
{

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);

    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();

    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime;
    return duration;
}

void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
